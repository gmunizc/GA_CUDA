#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <assert.h>

__global__ void randWork(unsigned int *seed, hiprandState_t* states, int *d_rnumbs)
{
   hiprand_init(seed[threadIdx.x],threadIdx.x,0,&states[threadIdx.x]);
   d_rnumbs[threadIdx.x] = hiprand(&states[threadIdx.x])% 100;

}

int main(){

  int nThreads = 10;

  hiprandState_t* states;
  unsigned int *h_seed = (unsigned int*)malloc(sizeof(unsigned int)*nThreads);
  srand(time(NULL));
  for(int i=0;i<nThreads;i++)
  {
    h_seed[i] = rand()%100000;
  }

  int *rnumbs = (int*)malloc(sizeof(int)*nThreads);
  int *d_rnumbs = (int*)malloc(sizeof(int)*nThreads);
  hipMalloc((void**)&d_rnumbs, sizeof(int)*nThreads);

  hipMalloc((void**) &states, nThreads * sizeof(hiprandState_t));
  unsigned int *d_seed;

  hipMalloc((void**)&d_seed, sizeof(unsigned int)*nThreads);
  hipMemcpy(d_seed, h_seed, sizeof(unsigned int)*nThreads,hipMemcpyHostToDevice);

  // veja somente parametros d_seed e states
  randWork<<<1,nThreads>>>(d_seed ,states,d_rnumbs);

  hipMemcpy(rnumbs, d_rnumbs, sizeof(int)*nThreads,hipMemcpyDeviceToHost);

  hipFree(states); hipFree(d_seed);
  free(h_seed);


  printf("Random Numbers:\n");
  for (int i = 0; i < nThreads; i++)
    printf("%d: %d\n", i, rnumbs[i]);

  return 0;
}
